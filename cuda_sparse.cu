
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <math.h>
#include <sys/time.h>
#include <unistd.h>
//#include "tools.h"

__global__ void matrixMultiply(int *rowvec, int *colvec, float *valvec, float *B, float *C, int I, int J, int K) {
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;
    
    float tmp = 0.0;

    if(row < I && col < J) {
      tmp = C[rowvec[row]*J+col];
      for(int i=0; i<K; i++) {
        tmp += valvec[row] * B[colvec[row]*J+col];
      }
      C[rowvec[row]*J+col] = tmp;
    }
}

int make_sparse_percent(float per, int rdim, int cdim, float *A) {
    long int maxnums, cnt, check;
    maxnums = (long int) (per * (float)rdim * (float)cdim);
    int *vals;
    vals = (int*) malloc(maxnums*sizeof(int));
    for(long int i = 0; i < maxnums; i++) {
        vals[i] = -1;
    }
    cnt = 0;
    while(cnt < maxnums) {
        long int num = rand() % (rdim * cdim + 1);
        check = 0;
        for(long int i = 0; i < cnt; i++) {
            if(vals[i] == num) {
                check = 1;
                break;
            }
        }
        if(check == 0) {
            vals[cnt] = num;
            cnt++;
        }
    }
    for(int i=0; i < rdim; i++) {
        for(int j=0; j < cdim; j++) {
            for(int k=0; k < cnt; k++) {
                if((i+1)*(j+1) == vals[k]) {
                    A[i*cdim+j] = 0.0;
                }
            }
        }
    }
    return rdim * cdim - maxnums + 1;
}

void make_sparse_matrix(int rdim, int cdim, int *rowval, int *colval, float *value, float *A) {
    int cnt = 0;
    for(int i=0; i < rdim; i++) {
        for(int j=0; j < cdim; j++) {
            if(A[i*cdim+j] != 0.0) {
                rowval[cnt] = i;
                colval[cnt] = j;
                value[cnt] = A[i*cdim+j];
                cnt++;
            }
        }
    }
}

double wctime() {
  // calculate wall time.
  struct timeval tv;
  gettimeofday(&tv, NULL); 
  return (tv.tv_sec + 1E-6 * tv.tv_usec);
}

int main(int argc, char *argv[]) {
    int idim = 200;
    int jdim = 400;
    int kdim = 1000;
    int i, j, k;
    long int newdim;
    double t1;
    float nops, per;
    float *A, *B, *C, *actualC, *Bg, *Cg, *valg;
    int *rowg, *colg;
    A = (float*) malloc(idim*kdim*sizeof(float));
    B = (float*) malloc(kdim*jdim*sizeof(float));
    C = (float*) malloc(idim*jdim*sizeof(float));
    actualC = (float*) malloc(idim*jdim*sizeof(float));

    for(i=0; i<idim; i++) {
      for(j=0; j<jdim; j++) {
        C[i*jdim+j] = 0.0;
	actualC[i*jdim+j] = 0.0;
      }
      for(k=0; k<kdim; k++) {
        A[i*kdim+k] = 0.0;
      }
    }
    for(k=0; k<kdim; k++) {
      for(j=0; j<jdim; j++) {
        B[k*jdim+j] = 0.0;
      }
    }

    for(i=0; i<idim; i++) {
      for(k=0; k<kdim; k++) {
        A[i*kdim+k] = (float)rand() / (float)RAND_MAX;
      }
    }
    for(k=0; k<kdim; k++) {
      for(j=0; j<jdim; j++) {
        B[k*jdim+j] = (float)rand() / (float)RAND_MAX;
      }
    }

    per = 0.3;
    printf("Running with %0.1f%% sparsity\n", per * 100);
    newdim = make_sparse_percent(per, idim, kdim, A);
    int *rowval, *colval;
    float *value;
    rowval = (int*) malloc(newdim*sizeof(int));
    colval = (int*) malloc(newdim*sizeof(int));
    value = (float*) malloc(newdim*sizeof(float));
    make_sparse_matrix(idim, kdim, rowval, colval, value, A);

    printf("A matrix sample: \n");
    for(i=0; i<2; i++) {
      printf("[ ");
      for(k=0; k<10; k++) {
        printf("%f ", A[i*kdim+k]);
      }
      printf("]\n");
    }
    printf("\nB matrix sample: \n");
    for(k=0; k<2; k++) {
      printf("[ ");
      for(j=0; j<10; j++) {
        printf("%f ", B[k*jdim+j]);
      }
      printf("]\n");
    }

    // This is the standard matrix multiplication - do not adjust
    for(i = 0; i < idim; i++) {
        for(k = 0; k < kdim; k++) {
            for(j = 0; j < jdim; j++) {
                actualC[i*jdim+j] += A[i*kdim+k] * B[k*jdim+j];
            }
        }
    }

    printf("\nActualC matrix sample: \n");
    for(i=0; i<2; i++) {
      printf("[ ");
      for(j=0; j<10; j++) {
        printf("%f ", actualC[i*jdim+j]);
      }
      printf("]\n");
    }
/*
//    #pragma omp parallel
    nt = omp_get_num_threads();
    printf("Running with %d threads\n", nt);
    t1 = wctime();
//    #pragma omp parallel for
    for(i = 0; i < idim; i++) {
        for(k = 0; k < kdim; k++) {
            for(j = 0; j < jdim; j++) {
                C[i*jdim+j] += A[i*kdim+k] * B[k*jdim+j];
            }
        }
    }
    t1 = wctime() - t1;
*/
    //cudaMalloc(&Ag, idim*kdim*sizeof(float));
    hipMalloc(&rowg, newdim*sizeof(int));
    hipMalloc(&colg, newdim*sizeof(int));
    hipMalloc(&valg, newdim*sizeof(float));
    hipMalloc(&Bg, kdim*jdim*sizeof(float));
    hipMalloc(&Cg, idim*jdim*sizeof(float));
    //cudaMalloc(&I, sizeof(int));
    //cudaMalloc(&J, sizeof(int));
    //cudaMalloc(&K, sizeof(int));
    t1 = wctime();
    //cudaMemcpy(Ag, A, idim*kdim*sizeof(float), cudaMemcpyHostToDevice);
    hipMemcpy(rowg, rowval, newdim*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(colg, colval, newdim*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(valg, value, newdim*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Bg, B, kdim*jdim*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Cg, C, idim*jdim*sizeof(float), hipMemcpyHostToDevice);
    //cudaMemcpy(I, idim, sizeof(int), cudaMemcpyHostToDevice);
    //cudaMemcpy(J, jdim, sizeof(int), cudaMemcpyHostToDevice);
    //cudaMemcpy(K, kdim, sizeof(int), cudaMemcpyHostToDevice);
    dim3 threadsPerBlock(jdim*idim, 2);
    dim3 blocksPerGrid(1, 1);
    if (idim * jdim > 512) {
      threadsPerBlock.x = 32;
      threadsPerBlock.y = 32;
      blocksPerGrid.x = ceil((double)jdim/(double)threadsPerBlock.x);
      blocksPerGrid.y = ceil((double)idim/(double)threadsPerBlock.y);
    }
    printf("threadsPerBlock: (%d, %d)\n", threadsPerBlock.x, threadsPerBlock.y);
    printf("blocksPerGrid:   (%d, %d)\n", blocksPerGrid.x, blocksPerGrid.y);
    //t1 = wctime();
    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(rowg, colg, valg, Bg, Cg, newdim, jdim, kdim);
    hipDeviceSynchronize();
    //t1 = wctime() - t1;
    hipError_t error = hipGetLastError();
    if (error) {
      printf("CUDA error: %s \n", hipGetErrorString(error));
      exit(1);
    }
    hipMemcpy(C, Cg, idim*jdim*sizeof(float), hipMemcpyDeviceToHost);
    t1 = wctime() - t1;
    // error calculation
    printf("\nC matrix sample: \n");
    for(i=0; i<2; i++) {
      printf("[ ");
      for(j=0; j<10; j++) {
        printf("%f ", C[i*jdim+j]);
      }
      printf("]\n");
    }
    float err = 0.0, t = 0.0;
    for(i = 0; i < idim; i++) {
        for(j = 0; j < jdim; j++) {
            err += ((actualC[i*jdim+j] - C[i*jdim+j]) * (actualC[i*jdim+j] - C[i*jdim+j]));
	    t += (actualC[i*jdim+j] * actualC[i*jdim+j]);
        }
    }
    err = sqrt(err/t);

    printf("Finished in %lf seconds\n", t1);
    t1 *= (1.e+09);
    nops = (float) 2 * idim * kdim * jdim;
    printf("Performance = %f GFLOPs\n",nops/t1);
    printf("Error: %f\n", err/((float)idim*jdim));
    //cudaFree(Ag);
    hipFree(rowg);
    hipFree(colg);
    hipFree(valg);
    hipFree(Bg);
    hipFree(Cg);
    //cudaFree(I);
    //cudaFree(J);
    //cudaFree(K);
    return(0);
}

