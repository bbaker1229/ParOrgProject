
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <math.h>
#include <sys/time.h>
#include <unistd.h>
//#include "tools.h"

__global__ void matrixMultiply(float *A, float *B, float *C, int I, int J, int K) {
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;
    float tmp = 0.0;
    if(row < I && col < J) {
      for(int i=0; i<K; i++) {
        tmp += A[row*K+i] * B[i*K+col];
      }
    }
    C[row*K+col] = tmp;
}

double wctime() {
  // calculate wall time.
  struct timeval tv;
  gettimeofday(&tv, NULL); 
  return (tv.tv_sec + 1E-6 * tv.tv_usec);
}

int main(int argc, char *argv[]) {
    int idim = 200;
    int jdim = 400;
    int kdim = 1000;
    int i, j, k;
    double t1;
    float nops;
    float *A, *B, *C, *actualC, *Ag, *Bg, *Cg;
    A = (float*) malloc(idim*kdim*sizeof(float));
    B = (float*) malloc(kdim*jdim*sizeof(float));
    C = (float*) malloc(idim*jdim*sizeof(float));
    actualC = (float*) malloc(idim*jdim*sizeof(float));

    for(i=0; i<idim; i++) {
      for(j=0; j<jdim; j++) {
        C[i*jdim+j] = 0.0;
	actualC[i*jdim+j] = 0.0;
      }
      for(k=0; k<kdim; k++) {
        A[i*kdim+k] = 0.0;
      }
    }
    for(k=0; k<kdim; k++) {
      for(j=0; j<jdim; j++) {
        B[k*jdim+j] = 0.0;
      }
    }

    // This is the standard matrix multiplication - do not adjust
    for(i = 0; i < idim; i++) {
        for(k = 0; k < kdim; k++) {
            for(j = 0; j < jdim; j++) {
                actualC[i*jdim+j] += A[i*kdim+k] * B[k*jdim+j];
            }
        }
    }
/*
//    #pragma omp parallel
    nt = omp_get_num_threads();
    printf("Running with %d threads\n", nt);
    t1 = wctime();
//    #pragma omp parallel for
    for(i = 0; i < idim; i++) {
        for(k = 0; k < kdim; k++) {
            for(j = 0; j < jdim; j++) {
                C[i*jdim+j] += A[i*kdim+k] * B[k*jdim+j];
            }
        }
    }
    t1 = wctime() - t1;
*/
    hipMalloc(&Ag, idim*kdim*sizeof(float));
    hipMalloc(&Bg, kdim*jdim*sizeof(float));
    hipMalloc(&Cg, idim*jdim*sizeof(float));
    //cudaMalloc(&I, sizeof(int));
    //cudaMalloc(&J, sizeof(int));
    //cudaMalloc(&K, sizeof(int));
    t1 = wctime();
    hipMemcpy(Ag, A, idim*kdim*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Bg, B, kdim*jdim*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Cg, C, idim*jdim*sizeof(float), hipMemcpyHostToDevice);
    //cudaMemcpy(I, idim, sizeof(int), cudaMemcpyHostToDevice);
    //cudaMemcpy(J, jdim, sizeof(int), cudaMemcpyHostToDevice);
    //cudaMemcpy(K, kdim, sizeof(int), cudaMemcpyHostToDevice);
    dim3 threadsPerBlock(kdim, kdim);
    dim3 blocksPerGrid(1, 1);
    //t1 = wctime();
    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(Ag, Bg, Cg, idim, jdim, kdim);
    hipDeviceSynchronize();
    //t1 = wctime() - t1;
    hipMemcpy(C, Cg, idim*jdim*sizeof(float), hipMemcpyDeviceToHost);
    t1 = wctime() - t1;
    // error calculation
    float err = 0.0;
    for(i = 0; i < idim; i++) {
        for(j = 0; j < jdim; j++) {
            err += (actualC[i*jdim+j] - C[i*jdim+j]);
        }
    }

    printf("Finished in %lf seconds\n", t1);
    t1 *= (1.e+09);
    nops = (float) 2 * idim * kdim * jdim;
    printf("Performance = %f GFLOPs\n",nops/t1);
    printf("Error: %f\n", err);
    hipFree(Ag);
    hipFree(Bg);
    hipFree(Cg);
    //cudaFree(I);
    //cudaFree(J);
    //cudaFree(K);
    return(0);
}
